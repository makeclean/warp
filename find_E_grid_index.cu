#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "datadef.h"

__global__ void find_E_grid_index_kernel(unsigned N, cross_section_data* d_xsdata , unsigned* remap, float* E , unsigned * index, unsigned* rxn){

	int tid_in = threadIdx.x+blockIdx.x*blockDim.x;
	if (tid_in >= N){return;}

	// declare shared variables			
	__shared__ 	unsigned			energy_grid_len;				
	__shared__ 	float*				energy_grid;			

	// have thread 0 of block copy all pointers and static info into shared memory
	if (threadIdx.x == 0){
		energy_grid_len				= d_xsdata[0].energy_grid_len;								
		energy_grid 				= d_xsdata[0].energy_grid;						
	}

	// make sure shared loads happen before anything else
	__syncthreads();

	// return if terminated
	unsigned this_rxn=rxn[tid_in];
	if (this_rxn>=900){return;}
	if (E[tid_in] <= 0.) {
	   rxn[tid_in] = 900;
	   return;
	}

	// remap
	int tid=remap[tid_in];

	// load data
	float value = E[tid];

	// init local
	unsigned cnt  = 0;
	unsigned powtwo = 2;
	int dex  = (energy_grid_len-1) / 2;  // N_energies starts at 1, duh

	// check edges
	float	grid_max = energy_grid[energy_grid_len-1];
	float	grid_min = energy_grid[0];
	if (value < grid_min){
		dex=-2;   // encoding for under min interpolation (2^32-2, or -2)
	}
	else if (value > grid_max){
		dex=-1;   // encoding for over max interpolation (2^32-1, or -1)
	}
	else{
		// do the search
		for(cnt=0;cnt<=30;cnt++){
			powtwo = powtwo * 2;
			if      ( 	energy_grid[dex]   <= value && 
					energy_grid[dex+1] >  value ) { break; }
			else if ( 	energy_grid[dex]   >  value ) { dex  = dex - ((energy_grid_len / powtwo) + 1) ;}  // +1's are to do a ceiling instead of a floor on integer division
			else if ( 	energy_grid[dex]   <  value ) { dex  = dex + ((energy_grid_len / powtwo) + 1) ;}
	
			if(cnt==30){
				printf("binary search iteration overflow! %p len %d val % 6.4E tid=%u rxn=%u\n",energy_grid,energy_grid_len,value,tid,this_rxn);
				dex=0;
			}
	
			// edge checks... fix later???
			if(dex<0){
				dex=0;
			}
			if(dex>=energy_grid_len){
				dex=energy_grid_len-1;
			}
		}
	}

	//write output index
	index[tid]=dex;

	//printf("rxn %u, remap[%i]=%u E %6.4E dex %u\n",this_rxn,tid_in,tid, value, dex);

}


/**
 * \brief a
 * \details b
 *
 * @param[in]    NUM_THREADS    - the number of threads to run per thread block
 * @param[in]    N              - the total number of threads to launch on the grid
 * @param[in]    d_xsdata       - device pointer to cross section data pointer array 
 * @param[in]    d_remap        - device pointer to data remapping vector
 * @param[in]    d_E            - device pointer to energy data array
 * @param[in]    d_index        - device pointer to index array (stores the unionized grid index of the current energy)
 * @param[in]    d_rxn          - device pointer of the reaction number array
 */ 
void find_E_grid_index(unsigned NUM_THREADS, unsigned N, cross_section_data* d_xsdata, unsigned* d_remap, float* d_E , unsigned * d_index , unsigned* d_rxn){

	unsigned blks = ( N + NUM_THREADS - 1 ) / NUM_THREADS;

	find_E_grid_index_kernel <<< blks, NUM_THREADS >>> ( N, d_xsdata, d_remap,  d_E , d_index , d_rxn);
	hipDeviceSynchronize();

}

